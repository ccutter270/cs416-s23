#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "CycleTimer.h"
#include "cuda-util.h"

/**
 * @brief Saxpy CUDA kernel
 * 
 * The __global__ keyword indicates that this is a device function that can be called from
 * the host via <<< ... >> kernel launch extension.
 * @param n Array length
 * @param alpha "A" in SAXPY
 * @param x Device "X" operand array
 * @param y Device "Y" operand array
 */
__global__
void SaxpyKernel(int n, float alpha, float* x, float* y) {
  // TODO: Implement the SAXPY kernel assuming 1-D thread blocks. Make sure to account for mismatch
  // between the total thread count and the size of the input
}

// The number of threads in the block (implemented here with BLOCK_SIZE) is often a
// fixed parameter. Here we specify it with a constant variable, you will also often 
// see preprocessor defines or template parameters.
static const int BLOCK_SIZE=256;

void SaxpyCUDA(int n, float alpha, float x[], float y[]) {
  // Define and allocate memory on device for input and output
  float* d_x;
  float* d_y;
  
  cudaErrorCheck( hipMalloc(&d_x, n*sizeof(float)) );
  cudaErrorCheck( hipMalloc(&d_y, n*sizeof(float)) );

  // Time total execution time, including the data transfers between host and device
  double start_time = CycleTimer::currentSeconds();
  
  // TODO: Copy the data from the host to the device with the appropriate cuda runtime call

  double kernel_start_time = CycleTimer::currentSeconds();
  
  // TODO: Implement the kernel launch. You will need to create enough blocks to compute
  // all n values (with blocks of BLOCK_SIZE threads)

  // Kernel launches are asynchronous. To ensure that we can accurately time the kernel execution
  // wait until all kernels are finished executing.
  cudaErrorCheck( hipDeviceSynchronize() );
  
  double kernel_end_time = CycleTimer::currentSeconds();

  // Copy array of results back to host from GPU
  cudaErrorCheck( hipMemcpy(y, d_y, n*sizeof(float), hipMemcpyDeviceToHost) );
  double end_time = CycleTimer::currentSeconds();

  double transfer_time = (end_time-start_time) - (kernel_end_time-kernel_start_time);
  double approx_device_bw = n * 3 * sizeof(float) / (kernel_end_time-kernel_start_time) / 1000 / 1000 / 1000;
  double approx_transfer_bw = n * 3 * sizeof(float) / transfer_time / 1000 / 1000 / 1000;

  printf("[saxpy cuda total]:\t%.3f ms\t%.3f GB/s host-device bandwidth\t%.3f GB/s device memory bandwidth\n", (end_time-start_time) * 1000, approx_transfer_bw, approx_device_bw);
  printf("[saxpy cuda kernel]:\t%.3f ms\n", (kernel_end_time-kernel_start_time) * 1000);
  printf("[saxpy cuda transfer]:\t%.3f ms\n", transfer_time * 1000);

  cudaErrorCheck( hipFree(d_x) );
  cudaErrorCheck( hipFree(d_y) );
}

void SaxpyCUDABLAS(int n, float alpha, float x[], float y[]) {
  float* d_x;
  float* d_y;

  hipMalloc(&d_x, n*sizeof(float));
  hipMalloc(&d_y, n*sizeof(float));

  // cublas has its own copy functions (in place of hipMemcpy)
  hipblasSetVector(n, sizeof(float), x, 1, d_x, 1);
  hipblasSetVector(n, sizeof(float), y, 1, d_y, 1);

  // Actually execute the cublas SAXPY kernel
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSaxpy(handle, n, &alpha, d_x, 1, d_y, 1);
  hipblasDestroy(handle);

  // Copy array of results back to host from GPU
  hipblasGetVector(n, sizeof(float), d_y, 1, y, 1);

  hipFree(d_x);
  hipFree(d_y);
}
