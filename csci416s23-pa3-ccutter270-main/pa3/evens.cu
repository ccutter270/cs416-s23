#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cub/block/block_reduce.cuh>
#include <cub/block/block_scan.cuh>
#include <cub/device/device_scan.cuh>
#include <cub/device/device_select.cuh>
#include <cub/iterator/counting_input_iterator.cuh>
#include <cub/iterator/transform_input_iterator.cuh>

#include "cuda-util.h"

#define SCAN_BLOCK_SIZE 512

__global__ void EvensCount(int n, int* values, int* block_counts) {
  // Specialize int BlockReduce for a 1D block of SCAN_BLOCK_SIZE threads
  typedef hipcub::BlockReduce<int, SCAN_BLOCK_SIZE> BlockReduce;

  // Allocate shared memory for BlockReduce
  __shared__ typename BlockReduce::TempStorage reduce_temp;

  // Load value for this thread
  int thread_idx = blockIdx.x * SCAN_BLOCK_SIZE + threadIdx.x;

  int thread_value = 1; // Make sure values past input don't contribute to the count
  if (thread_idx < n) {
    thread_value = values[thread_idx];
  }

  // Assign to be 1 if value is even
  int thread_is_even = static_cast<int>((thread_value % 2) == 0); 

  // Collectively compute the block-wide reduction to determine number of evens
  int total_evens = BlockReduce(reduce_temp).Sum(thread_is_even);

  if (threadIdx.x == 0)
    block_counts[blockIdx.x] = total_evens;
}

__global__ void EvensScatter(int n, int* values, int* block_boundaries, int* indices) {
  // Specialize int BlockScan for a 1D block of SCAN_BLOCK_SIZE threads
  typedef hipcub::BlockScan<int, SCAN_BLOCK_SIZE> BlockScan;

  // Allocate shared memory for BlockScan
  __shared__ typename BlockScan::TempStorage scan_temp;

  // Load value for this thread
  int thread_idx = blockIdx.x * SCAN_BLOCK_SIZE + threadIdx.x;
  
  int thread_value = 1; // Make sure values past input don't contribute to the count
  if (thread_idx < n) {
    thread_value = values[thread_idx];
  } 

  // Assign to be 1 if value is even
  int thread_is_even = static_cast<int>((thread_value % 2) == 0);

  // Since each thread has its own copy of local variables, each variable is
  // like an int[THREAD_COUNT] array. CUB uses local variables for input and output.
  int thread_scatter_index;

  // Collectively compute the block-wide exclusive prefix sum
  BlockScan(scan_temp).ExclusiveSum(thread_is_even, thread_scatter_index);

  // Scatter indices based on block-wise exclusive prefix sum
  int block_start = block_boundaries[blockIdx.x];
  if (thread_idx < n && thread_is_even) {
    indices[block_start + thread_scatter_index] = thread_idx;
  }
}

void EvensBlockCUB(int n, int values[], int indices[], int* indices_count) {
  int* d_values;
  int* d_indices;
  int* d_block_counts;
  int* d_block_boundaries;

  int blocks = (n + SCAN_BLOCK_SIZE - 1) / SCAN_BLOCK_SIZE;

  // Allocate device memory
  cudaErrorCheck(hipMalloc(&d_values, n * sizeof(int)));
  cudaErrorCheck(hipMalloc(&d_indices, n * sizeof(int)));
  cudaErrorCheck(hipMalloc(&d_block_counts, blocks * sizeof(int)));
  cudaErrorCheck(hipMalloc(&d_block_boundaries, (blocks + 1) * sizeof(int)));
  cudaErrorCheck(hipMemset(d_block_boundaries, 0, (blocks + 1) * sizeof(int)));
  
  // Transfer values to device
  cudaErrorCheck(hipMemcpy(d_values, values, n * sizeof(int), hipMemcpyHostToDevice));

  // 1. Compute counts of evens in each block
  EvensCount<<<blocks, SCAN_BLOCK_SIZE>>>(n, d_values, d_block_counts);

  // 2. Perform global prefix sum to determine global block boundaries

  // Determine temporary device storage requirements for scan
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_block_counts, d_block_boundaries + 1, blocks);

  // Allocate temporary storage, run device prefix sum, then free temporary storage
  cudaErrorCheck(hipMalloc(&d_temp_storage, temp_storage_bytes));
  hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_block_counts, d_block_boundaries + 1, blocks);
  cudaErrorCheck(hipFree(d_temp_storage));

  // 3. Perform block-wise scan and scatter to write out indices
  EvensScatter<<<blocks, SCAN_BLOCK_SIZE>>>(n, d_values, d_block_boundaries, d_indices);

  // Transfer indices back from device
  cudaErrorCheck(hipMemcpy(indices, d_indices, n*sizeof(int), hipMemcpyDeviceToHost));
  cudaErrorCheck(hipMemcpy(indices_count, d_block_boundaries+blocks, sizeof(int), hipMemcpyDeviceToHost));

  // Free device memory
  cudaErrorCheck(hipFree(d_values));
  cudaErrorCheck(hipFree(d_indices));
  cudaErrorCheck(hipFree(d_block_counts));
  cudaErrorCheck(hipFree(d_block_boundaries));
}

/**
 * @brief Functor for determining if a number is even
 *
 * A functor (function object) is a object that overloads the function call operator
 * so that it can be called like a function
 */
struct IsEven {
    __host__ __device__ __forceinline__
    bool operator()(const int& value) const {
        return (value % 2) == 0;
    }
};

void EvensDeviceCUB(int n, int values[], int indices[], int* indices_count) {
  int* d_values;
  int* d_indices;
  int* d_num_evens;

  cudaErrorCheck(hipMalloc(&d_values, n * sizeof(int)));
  cudaErrorCheck(hipMalloc(&d_indices, n * sizeof(int)));
  cudaErrorCheck(hipMalloc(&d_num_evens, sizeof(int)));

  cudaErrorCheck(hipMemcpy(d_values, values, n * sizeof(int), hipMemcpyHostToDevice));

  // Create an iterator wrapper for indices
  hipcub::CountingInputIterator<int> index_itr(0);
  
  // Create an iterator wrapper for computing "is even" from d_values
  IsEven is_even_op;
  hipcub::TransformInputIterator<bool, IsEven, int*> is_even_itr(d_values, is_even_op);

  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, index_itr, is_even_itr, d_indices, d_num_evens, n);
  
  // Allocate temporary storage, run selection, free temporary storage
  cudaErrorCheck(hipMalloc(&d_temp_storage, temp_storage_bytes));
  hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, index_itr, is_even_itr, d_indices, d_num_evens, n);
  cudaErrorCheck(hipFree(d_temp_storage));

  // Transfer indices back from device
  cudaErrorCheck(hipMemcpy(indices, d_indices, n*sizeof(int), hipMemcpyDeviceToHost));
  cudaErrorCheck(hipMemcpy(indices_count, d_num_evens, sizeof(int), hipMemcpyDeviceToHost));

  cudaErrorCheck(hipFree(d_values));
  cudaErrorCheck(hipFree(d_indices));
  cudaErrorCheck(hipFree(d_num_evens));
}