#include "hip/hip_runtime.h"
#include "render.h"
#include "cuda-util.h"


namespace {
  /**
   * @brief Return v clamped to be within the range (min, max)
   * 
   * If v is smaller than min, return min, if it is larger than max, return max, otherwise
   * return v.
   */
  template<typename T> __device__ __host__ inline 
  T clamp(T v, T min, T max) {
    return (v < min) ? min : ((v > max) ? max : v);
  }
  
  /// Return 1 if a circle of radius centered on x,y may overlap a rectangle. All arguments
  /// are in the normalized [0,1] coordinate space.
  ///
  /// This function is sensitive and may return 1 even if the circle does not
  /// overlap the rectangle but if it returns 0, the circle is guaranteed to not 
  /// overlap the rectangle.
  __device__ __inline__ int
  CircleOverlapsBoxFast(float x, float y, float radius, float left, float right, float top, float bottom) {
    // Adapted from CS149 routine
    if (x >= (left - radius) && x <= (right + radius) && 
        y >= (bottom - radius) && y <= (top + radius)) {
      return 1;
    } else {
      return 0;
    }
  }

  /// Return 1 if a circle of radius centered on x,y overlaps a rectangle. All arguments
  /// are in the normalized [0,1] coordinate space.
  ///
  /// This function is precise and will only return 1 if the circle overlaps
  /// the rectangle.
  __device__ __inline__ int
  CircleOverlapsBoxPrecise(float x, float y, float radius, float left, float right, float top, float bottom) {
    // Adapted from CS149 routine
    
    // Find the closest point in the box to the circle
    float closest_x = clamp(x, left, right);
    float closest_y = clamp(y, bottom, top);
    
    // Compute the distance between closest point and circle center. If that distance is less than 
    // radius then return 1.
    float delta_x = closest_x - x;
    float delta_y = closest_y - y;
    if ((delta_x * delta_x + delta_y * delta_y) <= (radius * radius)) {
      return 1;
    } else {
      return 0;
    }
  }
}


__global__ void RenderCUDAKernel(int n_circles, float2* circles_position, float* circles_radius, float4* circles_color, int image_width, int image_height, float4* image_data) {
  // Determine the circle to be computed by this thread
  int c = blockIdx.x * blockDim.x + threadIdx.x;

  // Compute the width and height of each pixel in normalized [0,1] coordinates
  float x_width = 1.f / image_width;
  float y_width = 1.f / image_height;

  float2 center = circles_position[c];
  float radius = circles_radius[c];
  float4 color = circles_color[c];
  
  // Compute the bounding box for the circle
  float min_x = center.x - radius;
  float max_x = center.x + radius;
  float min_y = center.y - radius;
  float max_y = center.y + radius;

  // Determine the pixels within the bounding box by translating normalized coordinate space
  // to pixel indexes
  int beg_x = clamp(static_cast<int>(min_x * image_width), 0, image_width);
  int end_x = clamp(static_cast<int>(max_x * image_width) + 1, 0, image_width);
  int beg_y = clamp(static_cast<int>(min_y * image_height), 0, image_height);
  int end_y = clamp(static_cast<int>(max_y * image_height) + 1, 0, image_height);

  // Iterate through all of the pixels in the bounding box
  for (int y=beg_y; y<end_y; y++) {
    for (int x=beg_x; x<end_x; x++) {
      // Determine pixel center (in the "middle" of the pixel) in normalized [0,1] coordinate space
      float pix_x = x_width * (static_cast<float>(x) + 0.5f);
      float pix_y = y_width * (static_cast<float>(y) + 0.5f);

      // Check that the pixel center is actually within the circle
      float delta_x = center.x - pix_x;
      float delta_y = center.y - pix_y;
      if ((delta_x * delta_x + delta_y * delta_y) > (radius * radius))
        continue; // Pixel is outside the circle

      // Obtain the actual pixel color data assuming row-major storage. Use a pointer
      // so that we can modify the underlying data. 
      float4* pixel_color = image_data + y*image_width + x;
      float alpha = color.w;

      // Update pixel colors with alpha blending. This must be an atomic
      // read-modify-write operation, and must be performed for each circle in 
      // correct order to produce the correct colors.
      pixel_color->x = alpha * color.x + (1 - alpha) * pixel_color->x;
      pixel_color->y = alpha * color.y + (1 - alpha) * pixel_color->y;
      pixel_color->z = alpha * color.z + (1 - alpha) * pixel_color->z;
      pixel_color->w = alpha + (1 - alpha) * pixel_color->w;
    }  
  }

}

// Do not modify this interface. All other modifications, such as to the interface of RenderCUDAKernel, are permitted.
void RenderCUDA(Image& image, const Circles& circles) {
  // Allocate device data
  float2* d_circles_position;
  float* d_circles_radius;
  float4* d_circles_color;
  float4* d_image_data;

  int n_circles = circles.n_;
  cudaErrorCheck( hipMalloc(&d_circles_position, n_circles*sizeof(float2)) );
  cudaErrorCheck( hipMalloc(&d_circles_radius, n_circles*sizeof(float)) );
  cudaErrorCheck( hipMalloc(&d_circles_color, n_circles*sizeof(float4)) );
  cudaErrorCheck( hipMalloc(&d_image_data, image.width_*image.height_*sizeof(float4)) );
  
  // Copy circle and image data to the device
  cudaErrorCheck( hipMemcpy(d_circles_position, circles.position_, n_circles*sizeof(float2), hipMemcpyHostToDevice) );
  cudaErrorCheck( hipMemcpy(d_circles_radius, circles.radius_, n_circles*sizeof(float), hipMemcpyHostToDevice) );
  cudaErrorCheck( hipMemcpy(d_circles_color, circles.color_, n_circles*sizeof(float4), hipMemcpyHostToDevice) );
  cudaErrorCheck( hipMemcpy(d_image_data, image.data_, image.width_*image.height_*sizeof(float4), hipMemcpyHostToDevice) );


  // Launch kernel with one block per circle, with one thread per-block
  RenderCUDAKernel<<<circles.n_, 1>>>(n_circles, d_circles_position, d_circles_radius, d_circles_color, image.width_, image.height_, d_image_data);

  // Copy rendered image back to host
  cudaErrorCheck( hipMemcpy(image.data_, d_image_data, image.width_*image.height_*sizeof(float4), hipMemcpyDeviceToHost) );

  // Cleanup device data
  cudaErrorCheck( hipFree(d_circles_position) );
  cudaErrorCheck( hipFree(d_circles_radius) );
  cudaErrorCheck( hipFree(d_circles_color) );
  cudaErrorCheck( hipFree(d_image_data) );
}
